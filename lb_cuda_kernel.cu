
#include <hip/hip_runtime.h>
#ifndef N
#define N 2160
#endif
#ifndef M
#define M 3840
#endif
#ifndef OMEGA
#define OMEGA 0.00000000000001f
#endif
#define PI 3.141592653589f

// Adapted from https://github.com/hellopatrick/cuda-samples/blob/master/hsv/kernel.cu
__device__ uchar3 hsv_to_rgb(float h, float s, float v) {
    float r, g, b;
    
    float f = h/(PI/3);
    float hi = floorf(f);
    f = f - hi;
    float p = v * (1 - s);
    float q = v * (1 - s * f);
    float t = v * (1 - s * (1 - f));
    
    if(hi == 0.0f || hi == 6.0f) {
        r = v;
        g = t;
        b = p;
    } else if(hi == 1.0f) {
        r = q;
        g = v;
        b = p;
    } else if(hi == 2.0f) {
        r = p;
        g = v;
        b = t;
    } else if(hi == 3.0f) {
        r = p;
        g = q;
        b = v;
    } else if(hi == 4.0f) {
        r = t;
        g = p;
        b = v;
    } else {
        r = v;
        g = p;
        b = q;
    }
    
    unsigned char red = (unsigned char) __float2uint_rn(255.0f * r);
    unsigned char green = (unsigned char) __float2uint_rn(255.0f * g);
    unsigned char blue = (unsigned char) __float2uint_rn(255.0f * b);
    return (uchar3) {red, green, blue};
}

// # Constants for D2Q9 https://arxiv.org/pdf/0908.4520.pdf # Normalized boltzmann distribution (thermal)
// assert(np.all(w == np.flip(w, axis=0)))
// assert(math.isclose(sum(w), 1, rel_tol=1e-6))
#define r2 3.0f
__constant__ const float w[3][3] = {{1.0/36, 1.0/9, 1.0/36},
                                    {1.0/9,  4.0/9, 1.0/9},
                                    {1.0/36, 1.0/9, 1.0/36}};

template<typename T>
struct cell_t {
    T d[3][3];
};

template<typename T>
struct grid_t {
    T d[N][M];
};



extern "C" {
__global__ void fused_collide_stream(grid_t<cell_t<float>>* newcells, grid_t<uchar3>* frame, const grid_t<cell_t<float>>* cells,
                                     const grid_t<bool>* blocked, const cell_t<float>* surroundings) {
    //assert(gridDim.z * blockDim.z == 1);
    //assert(gridDim.y * blockDim.y == N);
    //assert(gridDim.x * blockDim.x == M);

    //int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    bool isHalo = (x%32 == 0 || x%32 == 31);
    x -= (x/32) * 2 + 1;
    bool isEdge = (x < 0 || x >= M);
    isHalo = isEdge || isHalo;

    cell_t<float> surr = *surroundings;
    cell_t<float> prev = surr, curr = surr, next, newcurr;

    for(int y = 0; y < N + 1; y++) {

    // cell_t<float> asdf;
    // asdf.d[0][0] = 0; asdf.d[0][1] = 0; asdf.d[0][2] = 0;
    // asdf.d[1][0] = 0; asdf.d[1][1] = 0; asdf.d[1][2] = 100;
    // asdf.d[2][0] = 0; asdf.d[2][1] = 0; asdf.d[2][2] = 0;

    // Calculate aggregates
    if(isEdge || y >= N)
        next = surr;
    else
        next = cells->d[y][x];
    float s1 = next.d[0][0] + next.d[0][1] + next.d[0][2];
    float s2 = next.d[1][0] + next.d[1][1] + next.d[1][2];
    float s3 = next.d[2][0] + next.d[2][1] + next.d[2][2];
    float d = s1 + s2 + s3 + 0.0001; // Total density (plus a fudge factor for numerical stability)
                            // Alternative numerical stability method is to prevent any values from going negative.
    float uy = (s3 - s1)/d; // Y component of average velocity
    float ux = (next.d[0][2] + next.d[1][2] + next.d[2][2] - next.d[0][0] - next.d[1][0] - next.d[2][0])/d; // X component of average velocity
    // float mag = uy*uy + ux*ux;
    // uy /= mag;
    // ux /= mag;

    // Display the frame
    if (frame && !isHalo && y > 0) {
        float h = atan2f(uy, ux) + PI;
        float s = __saturatef(1000 * sqrtf(ux*ux+uy*uy));
        float v = __saturatef(d);
        frame->d[y-1][x] = hsv_to_rgb(h, s, v);
    }

    // Compute collide
    if(y > 0 && !blocked->d[y-1][x+1]) {
        float c = 1 - r2/2*(ux*ux + uy*uy);
        #pragma unroll
        for(int dy = 0; dy <= 2; dy ++) {
            #pragma unroll
            for(int dx = 0; dx <= 2; dx ++) {
                float eu = (dy-1) * uy + (dx-1) * ux;
                float eq = d * w[dy][dx] * (c + r2 * eu * (1 + r2/2*eu));
                next.d[dy][dx] = (next.d[dy][dx] - eq) * OMEGA + eq;
            }
        }
    }

    // Exchange adjacent through shuffles
    newcurr.d[0][0] = __shfl_down_sync(0xffffffff, next.d[0][0], 1);
    newcurr.d[0][1] = __shfl_down_sync(0xffffffff, curr.d[0][1], 1);
    newcurr.d[0][2] = __shfl_down_sync(0xffffffff, prev.d[0][2], 1);
    newcurr.d[1][0] = next.d[1][0];
    newcurr.d[1][1] = curr.d[1][1];
    newcurr.d[1][2] = prev.d[1][2];
    newcurr.d[2][0] = __shfl_up_sync(0xffffffff, next.d[2][0], 1);
    newcurr.d[2][1] = __shfl_up_sync(0xffffffff, curr.d[2][1], 1);
    newcurr.d[2][2] = __shfl_up_sync(0xffffffff, prev.d[2][2], 1);

    if(y > 0 && blocked->d[y-1][x]) {
        {
            float tmp = newcurr.d[0][0];
            newcurr.d[0][0] = newcurr.d[2][2];
            newcurr.d[2][2] = tmp;
        }
        {
            float tmp = newcurr.d[0][1];
            newcurr.d[0][1] = newcurr.d[2][1];
            newcurr.d[2][1] = tmp;
        }
        {
            float tmp = newcurr.d[0][2];
            newcurr.d[0][2] = newcurr.d[2][0];
            newcurr.d[2][0] = tmp;
        }
        {
            float tmp = newcurr.d[1][0];
            newcurr.d[1][0] = newcurr.d[1][2];
            newcurr.d[1][2] = tmp;
        }
    }

    if(!isHalo && x >= 0 && x < M && y > 0) {
        newcells->d[y-1][x] = newcurr; // Geez this is 100% of the gap between where we are (2150us) and memory bandwidth bottleneck (500us)
    }
    prev = curr;
    curr = next;
    }
}
}

