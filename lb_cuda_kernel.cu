#include "hip/hip_runtime.h"
#ifndef N
#define N 2160
#endif
#ifndef M
#define M 3840
#endif
#ifndef INNER_TIMESTEPS
#define INNER_TIMESTEPS 1
#endif
#ifndef INNER_BLOCK
#define INNER_BLOCK 2000000
#endif
#ifndef OMEGA
#define OMEGA 0.00000000000001f
#endif
#define PI 3.141592653589f

#include "hip/hip_fp16.h"

#ifdef half_enable
#define FP half
#else
#define FP float
#endif

// Adapted from https://github.com/hellopatrick/cuda-samples/blob/master/hsv/kernel.cu
__device__ uchar3 hsv_to_rgb(float h, float s, float v) {
    float r, g, b;
    
    float f = h/(PI/3);
    float hi = floorf(f);
    f = f - hi;
    float p = v * (1 - s);
    float q = v * (1 - s * f);
    float t = v * (1 - s * (1 - f));
    
    if(hi == 0.0f || hi == 6.0f) {
        r = v;
        g = t;
        b = p;
    } else if(hi == 1.0f) {
        r = q;
        g = v;
        b = p;
    } else if(hi == 2.0f) {
        r = p;
        g = v;
        b = t;
    } else if(hi == 3.0f) {
        r = p;
        g = q;
        b = v;
    } else if(hi == 4.0f) {
        r = t;
        g = p;
        b = v;
    } else {
        r = v;
        g = p;
        b = q;
    }
    
    unsigned char red = (unsigned char) __float2uint_rn(255.0f * r);
    unsigned char green = (unsigned char) __float2uint_rn(255.0f * g);
    unsigned char blue = (unsigned char) __float2uint_rn(255.0f * b);
    return (uchar3) {red, green, blue};
}

// # Constants for D2Q9 https://arxiv.org/pdf/0908.4520.pdf # Normalized boltzmann distribution (thermal)
// assert(np.all(w == np.flip(w, axis=0)))
// assert(math.isclose(sum(w), 1, rel_tol=1e-6))
#define r2 3.0f
__constant__ const float w[3][3] = {{1.0/36, 1.0/9, 1.0/36},
                                    {1.0/9,  4.0/9, 1.0/9},
                                    {1.0/36, 1.0/9, 1.0/36}};

template<typename T>
struct cell_t {
    T d[3][3];
};

template<typename T>
struct grid_t {
    T d[N][M];
};

template<typename T>
__device__ __forceinline__ void swap(T &a, T &b) {
    T tmp = a;
    a = b;
    b = tmp;
}

__device__ __forceinline__ void prefetch_l1 (unsigned int addr)
{
  asm volatile(" prefetch.global.L1 [ %1 ];": "=r"(addr) : "r"(addr));
}

__device__ __forceinline__ void prefetch_l2 (unsigned int addr)
{
  asm volatile(" prefetch.global.L2 [ %1 ];": "=r"(addr) : "r"(addr));
}

#ifdef USE_PACKED_CELLS
// Something to try......
// each cell is 9 * 4 bytes, which doesn't align well (you can load up to 16B in one insn)
// and also probably has a lot of shared info (i.e. shared-exponent floating point).
// This means that a custom binary format (instead of cell_t<half>) could be used for global memory,
// and then manually shifted into IEEE fp32 by h2f.
    // 8b exponent + 20b mantissa * 9 = 188b < 192b = 24B. 1 cell = 3 * LDG.E.64
    // ... could also just use fixed point (i.e. manipulate integers and assume some fixed exponent like 2**-8) lol
// This would necessitate more subtle comparison testing because it won't be as obviously wrong as fp16 was (which has 10 bits of mantissa).
// Would also be worth doing a validation against an analytical solution.

struct packed_cell {
    int2 d[3];
};

__device__ __forceinline__ cell_t<float> p2f(const packed_cell& p) {
    // assumes the sign is always positive... which it should be.
    cell_t<float> c;
    int mantissa = ((*(int*)&p.d[0]) & 255) << 23;
    return c;
}
__device__ __forceinline__ packed_cell f2p(const cell_t<float>& c) {
    // extract all the components
    // take the maximum exponent we can find
    // for each of the 9 components, rightshift the mantissa by (max - mine) to make it fixed point
    // then rightshift it again by 3 for the known precision reduction to 20b, which is still way better than the 10b of fp16

    packed_cell p;
    int maxexp = 0;

    for(int i = 0; i < 3; i++) {
        for(int j = 0; j < 3; j++) {
            int exp1 = ((*(int*)&c.d[i][j]) >> 23) & 255;
            maxexp = max(maxexp, exp1);
        }
    }
    copy_to_bit_offset(&packed_cell, maxexp, 10);
    for(int i = 0; i < 3; i++) {
        for(int j = 0; j < 3; j++) {
            int exp1 = ((*(int*)&c.d[i][j]) >> 23) & 255;
            int mantissa = ((*(int*)&c.d[i][j]) & ((1 << 23) - 1)) >> (maxexp - exp1 + 3);
        }
    }
    ((unsigned char*)&packed_cell)
    return p;
}
#endif // USE_PACKED_CELLS

#ifdef half_enable
__device__ __forceinline__ cell_t<float> h2f(const cell_t<half>& c) {
    cell_t<float> c2;
    #pragma unroll
    for(int i = 0; i <= 2; i ++) {
        #pragma unroll
        for(int j = 0; j <= 2; j ++) {
            c2.d[i][j] = c.d[i][j];
        }
    }
    return c2;
}

__device__ __forceinline__ cell_t<half> f2h(const cell_t<float>& c) {
    cell_t<half> c2;
    #pragma unroll
    for(int i = 0; i <= 2; i ++) {
        #pragma unroll
        for(int j = 0; j <= 2; j ++) {
            c2.d[i][j] = c.d[i][j];
        }
    }
    return c2;
}
#else
#define h2f(x) (x)
#define f2h(x) (x)
#endif

// Did not work :(
__device__ __forceinline__ cell_t<float> __ldcs_cell(const cell_t<float>* cell) {
    cell_t<float> c;
    // also tried ldg, ldca
    c.d[0][0] = __ldcs(((float*)cell) + 0);
    c.d[0][1] = __ldcs(((float*)cell) + 1);
    c.d[0][2] = __ldcs(((float*)cell) + 2);
    c.d[1][0] = __ldcs(((float*)cell) + 3);
    c.d[1][1] = __ldcs(((float*)cell) + 4);
    c.d[1][2] = __ldcs(((float*)cell) + 5);
    c.d[2][0] = __ldcs(((float*)cell) + 6);
    c.d[2][1] = __ldcs(((float*)cell) + 7);
    c.d[2][2] = __ldcs(((float*)cell) + 8);
    return c;
}

__device__ __forceinline__ void __stcs_cell(const cell_t<float>* cell, const cell_t<float>& c) {
    __stcs(((float*)cell) + 0, c.d[0][0]);
    __stcs(((float*)cell) + 1, c.d[0][1]);
    __stcs(((float*)cell) + 2, c.d[0][2]);
    __stcs(((float*)cell) + 3, c.d[1][0]);
    __stcs(((float*)cell) + 4, c.d[1][1]);
    __stcs(((float*)cell) + 5, c.d[1][2]);
    __stcs(((float*)cell) + 6, c.d[2][0]);
    __stcs(((float*)cell) + 7, c.d[2][1]);
    __stcs(((float*)cell) + 8, c.d[2][2]);
}

template<bool shouldDisplay>
__device__ void fcs(grid_t<cell_t<FP>>* newcells, grid_t<uchar3>* frame, const grid_t<cell_t<FP>>* cells,
                                     const grid_t<bool>* blocked, const cell_t<FP>* surroundings) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    bool isHalo = (x%32 < INNER_TIMESTEPS || x%32 > 31 - INNER_TIMESTEPS);
    x -= (x/32) * INNER_TIMESTEPS * 2 + INNER_TIMESTEPS;
    bool isEdge = (x < 0 || x >= M);
    isHalo = isEdge || isHalo;

    cell_t<float> surr = h2f(*surroundings);
    cell_t<float> prev, curr, next;

    // This fails because there is far too little shared memory and also I think we're already using L1/L2
    // (at the same INNER_BLOCK and INNER_TIMESTEPS it's actually slower to use shared than just directly accessing newcells[][]).
    // __shared__ cell_t<FP> cache[32][INNER_BLOCK + 2 * INNER_TIMESTEPS + 1];
    // If it's already making use of L1/L2, why is it so slow then? Especially on the multi-iteration runs.

    for(int z = -1; z < N; z += INNER_BLOCK) {
    if(z == -1) {
        prev = surr;
        curr = surr;
    } else {
        prev = cells->d[z-1][x]; // technically incorrect if you're incrementing z by 1 in the big loop
        curr = cells->d[z][x];
    }

    for(int timestep = 0; timestep < INNER_TIMESTEPS; timestep ++) {

    for(int y = z; y < z + INNER_BLOCK + 2 * INNER_TIMESTEPS && y < N; y++) {
        // if(y&31==0)
        //     prefetch_l2((unsigned int)&cells->d[y+32][x]); // This produces no appreciable benefit. :(
        // also tried using "nextnext" to get it loaded into registers on the previous iteration, but that didn't seem to help
        // why wouldn't it hit memory bandwidth then??? is it actually blocked by the latency (32ish cycles?) of loads from l1 to registers?
        // it could be legitimately compute bottlenecked... but that seems so unlikely given that other sims were able to hit mem bandwidth. Diff arch tho.
        // no, core overclocking does literally nothing and mem overclocking is basically linear.
        // ah possibly because it's unaligned / not using LDG.E.128.SYS aligned vectorized loads!
        // An attempt was previously made to simply expand the struct to 4 floats per row to use only aligned loads but that is -40% performance.
        // Could possibly interleave 9 128-bit loads, in 1024-bit coalesced chunks, to do four rows at once.
        // 1111 ... x32
        // 1111
        // 1222
        // 2222
        // 2233
        // 3333
        // 3334
        // 4444
        // 4444
        // so every memory load instruction is 1) LDG.E.128.SYS (float4) 2) aligned 3) coalesced
        // struct coalescing_block {
        //     float4 a[32], b[32], c[32], d[32], e[32], f[32], g[32], h[32], i[32];
        // }
        // Actually a better way to do this might be to have 9 separate arrays (SoA), each containing the direction, and load 4 at a time from each of those.
            // Register pressure though... 36 registers just from those loads. Maybe SoA is not the right way to go.
        // It's possible that the above doesn't increase register pressure if done carefully (only 3 128bit registers in use at a time, reuse the earlier registers) but that takes a lot of work.
        // I should probably just implement 3D lattice boltzmann first before attempting this micro-optimization, since we're already at 92% achievable.
        //
        // NOTE: According to the Occupancy Calculator the ideal register use is < 64 per thread. Since we are currently at 57, using any more will affect maximum occupancy.
        // An attempt to get some of the useless elements of prev out of registers didn't do anything:
            // prev0 = curr.d[0][2];
            // prev1 = curr.d[1][2];
            // prev2 = curr.d[2][2];
        //
        // NOTE 2: to circumvent the memory bottleneck, we need to do more compute per unit of data. That means maybe do 2 or more iterations of LBM on a given block. This increases halo size though.
            // But this doesn't have to be at the individual warp level - we can have larger chunks that fit into (half of?) L2 cache, which has 5x bandwidth, and iterate enough times (5?) to balance with global memory.
            // Just implement 3D first though, because all these considerations and tunings dramatically change.
        // NOTE 3: can we make it 32-wide instead of 30-wide? because 30-wide hobbles 3D. but the edge ones will have to read extra, stalling the others...
        // NOTE 4: register reuse caches and register banks/ports - see https://developer.download.nvidia.com/video/gputechconf/gtc/2019/presentation/s9839-discovering-the-turing-t4-gpu-architecture-with-microbenchmarks.pdf manual sass tuning

        // 50ish rows fit into L1
        // 80ish rows fit into L2
        // so lookahead to 20 while loading the next 20?? just test out some lookaheads for the inner inner loop.

        // Calculate aggregates
        if(isEdge || y < -1 || y >= N - 1)
            next = surr;
        else if(timestep == 0)
            next = h2f(cells->d[y+1][x]);
            // next = h2f(__ldcs_cell(&cells->d[y+1][x]));
        else
            next = h2f(newcells->d[y+1][x]);
        float s1 = next.d[0][0] + next.d[0][1] + next.d[0][2];
        float s2 = next.d[1][0] + next.d[1][1] + next.d[1][2];
        float s3 = next.d[2][0] + next.d[2][1] + next.d[2][2];
        float d = s1 + s2 + s3 + 0.0001; // Total density (plus a fudge factor for numerical stability)
                                         // Alternative numerical stability method is to prevent any values from going negative, or otherwise normalize.
        // Adding 10 floating point multiplies here kills performance by ~50%
        float uy = (s3 - s1)/d; // Y component of average velocity
        float ux = (next.d[0][2] + next.d[1][2] + next.d[2][2] - next.d[0][0] - next.d[1][0] - next.d[2][0])/d; // X component of average velocity

        if(y == -1) {
            prev = curr;
            curr = next;
            continue;
        }

        if constexpr(shouldDisplay) {
            // Display the frame
            if (timestep == INNER_TIMESTEPS - 1 && frame && !isHalo) {
                float h = atan2f(uy, ux) + PI;
                float s = __saturatef(1000 * sqrtf(ux*ux+uy*uy));
                float v = __saturatef(d);
                if(blocked->d[y][x]) {
                    h = s = v = 0;
                }
                frame->d[y][x] = hsv_to_rgb(h, s, v);
            }
        }

        // Compute collide (if blocked, don't do this - no thermal perturbation, purely reflection)
        if(!blocked->d[y][x]) {
            float c = 1 - r2/2*(ux*ux + uy*uy);
            #pragma unroll
            for(int dy = 0; dy <= 2; dy ++) {
                #pragma unroll
                for(int dx = 0; dx <= 2; dx ++) {
                    float eu = (dy-1) * uy + (dx-1) * ux;
                    float eq = d * w[dy][dx] * (c + r2 * eu * (1 + r2/2*eu));
                    next.d[dy][dx] = (next.d[dy][dx] - eq) * OMEGA + eq;
                }
            }
        }

        // Exchange adjacent through shuffles
        cell_t<float> newcurr;
        newcurr.d[0][0] = __shfl_down_sync(0xffffffff, next.d[0][0], 1);
        newcurr.d[0][1] = __shfl_down_sync(0xffffffff, curr.d[0][1], 1);
        newcurr.d[0][2] = __shfl_down_sync(0xffffffff, prev.d[0][2], 1);
        newcurr.d[1][0] = next.d[1][0];
        newcurr.d[1][1] = curr.d[1][1];
        newcurr.d[1][2] = prev.d[1][2];
        newcurr.d[2][0] = __shfl_up_sync(0xffffffff, next.d[2][0], 1);
        newcurr.d[2][1] = __shfl_up_sync(0xffffffff, curr.d[2][1], 1);
        newcurr.d[2][2] = __shfl_up_sync(0xffffffff, prev.d[2][2], 1);

        prev = curr;
        curr = next;

        if(!isEdge) {
            // Reflect the new cell if blocked
            if(blocked->d[y][x]) {
                swap(newcurr.d[0][0], newcurr.d[2][2]);
                swap(newcurr.d[0][1], newcurr.d[2][1]);
                swap(newcurr.d[0][2], newcurr.d[2][0]);
                swap(newcurr.d[1][0], newcurr.d[1][2]);
            }

            // Write the new cell if not a halo cell
            if(!isHalo && (y > z + INNER_TIMESTEPS || z == -1)) {
                newcells->d[y][x] = f2h(newcurr);
                // __stcs_cell(&newcells->d[y][x], f2h(newcurr));
            }
        }
    }

    }

    }
}
extern "C" {
    __global__ void fused_collide_stream(grid_t<cell_t<FP>>* newcells, grid_t<uchar3>* frame, const grid_t<cell_t<FP>>* cells,
        const grid_t<bool>* blocked, const cell_t<FP>* surroundings) {
        if(frame)
            fcs<true>(newcells, frame, cells, blocked, surroundings);
        else
            fcs<false>(newcells, frame, cells, blocked, surroundings);
    }
}

